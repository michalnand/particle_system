#include "hip/hip_runtime.h"
#include "particles_kernel.h"

__global__
void particles_main_gpu_kernel(unsigned int count, sParticle *particle, float dt)
{
  unsigned int idx  = threadIdx.x + blockIdx.x*blockDim.x;

  if (idx < count)
  {
    unsigned int other = 0;
    float distance_min = 1000000000.0;

    for (unsigned int i = 0; i < count; i++)
      if (i != idx)
      {
        float distance = 0.0;

        distance+= (particle[idx].x - particle[i].x)*(particle[idx].x - particle[i].x);
        distance+= (particle[idx].y - particle[i].y)*(particle[idx].y - particle[i].y);
        distance+= (particle[idx].z - particle[i].z)*(particle[idx].z - particle[i].z);

        distance = sqrt(distance);

        if (distance < distance_min)
        {
          distance_min = distance;
          other = i;
        }
      }

    if (distance_min < particle[idx].r+particle[other].r)
    {
      /*
      float m = 2.0*particle[other].m/(particle[idx].m + particle[other].m);
      float d = distance_min*distance_min;

      particle[idx].ax = -m*(particle[idx].vx - particle[other].vx)*(particle[idx].x - particle[other].x)*(particle[idx].x - particle[other].x)/d;
      particle[idx].ay = -m*(particle[idx].vy - particle[other].vy)*(particle[idx].y - particle[other].y)*(particle[idx].y - particle[other].y)/d;
      particle[idx].az = -m*(particle[idx].vz - particle[other].vz)*(particle[idx].z - particle[other].z)*(particle[idx].z - particle[other].z)/d;
      */

      particle[idx].vx*= -1.0;
      particle[idx].vy*= -1.0;
      particle[idx].vz*= -1.0;

    }
    else
    {
      particle[idx].ax = 0.0;
      particle[idx].ay = 0.0;
      particle[idx].az = 0.0;
    }
  }
}

__global__
void particles_update_gpu_kernel(unsigned int count, sParticle *particle, float dt)
{
  unsigned int idx  = threadIdx.x + blockIdx.x*blockDim.x;

  if (idx < count)
  {
    particle[idx].vx+= particle[idx].ax;
    particle[idx].vy+= particle[idx].ay;
    particle[idx].vz+= particle[idx].az;

    particle[idx].x+= particle[idx].vx*dt;
    particle[idx].y+= particle[idx].vy*dt;
    particle[idx].z+= particle[idx].vz*dt;
  }
}

__global__
void particles_limit_ranges_gpu_kernel(unsigned int count, sParticle *particle, float dt)
{
  unsigned int idx  = threadIdx.x + blockIdx.x*blockDim.x;

  if (idx < count)
  {
    float k = 0.999;

    if (particle[idx].x > 1.0)
    {
      particle[idx].x = k;
      particle[idx].vx*= -1.0;
    }
    else
    if (particle[idx].x < -1.0)
    {
      particle[idx].x = -k;
      particle[idx].vx*= -1.0;
    }
    else
    if (particle[idx].y > 1.0)
    {
      particle[idx].y = k;
      particle[idx].vy*= -1.0;
    }
    else
    if (particle[idx].y < -1.0)
    {
      particle[idx].y = -k;
      particle[idx].vy*= -1.0;
    }
    if (particle[idx].z > 1.0)
    {
      particle[idx].z = k;
      particle[idx].vz*= -1.0;
    }
    else
    if (particle[idx].z < -1.0)
    {
      particle[idx].z = -k;
      particle[idx].vz*= -1.0;
    }
  }
}







void particles_main_cpu_kernel(unsigned int idx, unsigned int count, sParticle *particle, float dt)
{
  unsigned int other = 0;
  float distance_min = 1000000000.0;

  for (unsigned int i = 0; i < count; i++)
    if (i != idx)
    {
      float distance = 0.0;

      distance+= (particle[idx].x - particle[i].x)*(particle[idx].x - particle[i].x);
      distance+= (particle[idx].y - particle[i].y)*(particle[idx].y - particle[i].y);
      distance+= (particle[idx].z - particle[i].z)*(particle[idx].z - particle[i].z);

      distance = sqrt(distance);

      if (distance < distance_min)
      {
        distance_min = distance;
        other = i;
      }
    }

    if (distance_min < particle[idx].r+particle[other].r)
    {
      /*
      float m = 2.0*particle[other].m/(particle[idx].m + particle[other].m);
      float d = distance_min*distance_min;

      particle[idx].ax = -m*(particle[idx].vx - particle[other].vx)*(particle[idx].x - particle[other].x)*(particle[idx].x - particle[other].x)/d;
      particle[idx].ay = -m*(particle[idx].vy - particle[other].vy)*(particle[idx].y - particle[other].y)*(particle[idx].y - particle[other].y)/d;
      particle[idx].az = -m*(particle[idx].vz - particle[other].vz)*(particle[idx].z - particle[other].z)*(particle[idx].z - particle[other].z)/d;
      */

      particle[idx].vx*= -1.0;
      particle[idx].vy*= -1.0;
      particle[idx].vz*= -1.0;
      
    }
    else
    {
      particle[idx].ax = 0.0;
      particle[idx].ay = 0.0;
      particle[idx].az = 0.0;
    }
}


void particles_update_cpu_kernel(unsigned int idx, sParticle *particle, float dt)
{
    particle[idx].vx+= particle[idx].ax;
    particle[idx].vy+= particle[idx].ay;
    particle[idx].vz+= particle[idx].az;

    particle[idx].x+= particle[idx].vx*dt;
    particle[idx].y+= particle[idx].vy*dt;
    particle[idx].z+= particle[idx].vz*dt;
}


void particles_limit_ranges_cpu_kernel(unsigned int idx, sParticle *particle, float dt)
{
    float k = 0.999;

    if (particle[idx].x > 1.0)
    {
      particle[idx].x = k;
      particle[idx].vx*= -1.0;
    }
    else
    if (particle[idx].x < -1.0)
    {
      particle[idx].x = -k;
      particle[idx].vx*= -1.0;
    }
    else
    if (particle[idx].y > 1.0)
    {
      particle[idx].y = k;
      particle[idx].vy*= -1.0;
    }
    else
    if (particle[idx].y < -1.0)
    {
      particle[idx].y = -k;
      particle[idx].vy*= -1.0;
    }
    if (particle[idx].z > 1.0)
    {
      particle[idx].z = k;
      particle[idx].vz*= -1.0;
    }
    else
    if (particle[idx].z < -1.0)
    {
      particle[idx].z = -k;
      particle[idx].vz*= -1.0;
    }
}
