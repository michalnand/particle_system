#include "hip/hip_runtime.h"
#include "particles.h"
#include "particles_kernel.h"

#define PARTICLES_USE_CUDA  1


Particles::Particles()
{
  particles = nullptr;
  m_count = 0;
  m_dt = 1;
  m_iteration = 0;
  m_size = 0.0;
}

Particles::Particles(unsigned int count, float size, float dt)
{
  particles = nullptr;
  m_count = 0;
  m_dt = 1;
  m_iteration = 0;
  m_size = 0.0;

  init(count, size, dt);
}

Particles::~Particles()
{
  uninint();
}


void Particles::init(unsigned int count, float size, float dt)
{
  uninint();
  this->m_count = count;
  this->m_dt = dt;
  this->m_size = size;

  m_iteration = 0;

  #ifdef PARTICLES_USE_CUDA
    hipMalloc(&particles, m_count*sizeof(sParticle));
  #else
    particles = new sParticle[m_count];
  #endif

  set_random();
}

void Particles::uninint()
{
  if (particles != nullptr)
  {
    #ifdef PARTICLES_USE_CUDA
      hipFree(particles);
    #else
      delete particles;
    #endif

    particles = nullptr;
    m_count = 0;
  }
}

sParticle Particles::get(unsigned int idx)
{
  sParticle result;

  #ifdef PARTICLES_USE_CUDA
    hipMemcpy(&result, &particles[idx], sizeof(sParticle), hipMemcpyDeviceToHost);
  #else
    result = particles[idx];
  #endif

  return result;
}

std::vector<sParticle> Particles::get_all()
{
  std::vector<sParticle> result;
  for (unsigned int i = 0; i < m_count; i++)
    result.push_back(get(i));

  return result;
}

std::string Particles::get_string(unsigned int idx)
{
  sParticle tmp = get(idx);
  std::string result;

  result+= std::to_string(tmp.x) + " ";
  result+= std::to_string(tmp.y) + " ";
  result+= std::to_string(tmp.z) + " ";

  result+= std::to_string(tmp.vx) + " ";
  result+= std::to_string(tmp.vy) + " ";
  result+= std::to_string(tmp.vz) + " ";

  result+= std::to_string(tmp.ax) + " ";
  result+= std::to_string(tmp.ay) + " ";
  result+= std::to_string(tmp.az) + " ";

  result+= std::to_string(tmp.m) + " ";
  result+= std::to_string(tmp.r) + " ";

  return result;
}

void Particles::print(unsigned int idx)
{
  std::string tmp = get_string(idx);
  printf("%s\n", tmp.c_str());
}

void Particles::process()
{
  #ifdef PARTICLES_USE_CUDA

    dim3 block(256);
    dim3 grid((m_count  + block.x - 1)/block.x);

    particles_main_gpu_kernel<<<grid, block>>>(m_count, particles, m_dt);
    hipDeviceSynchronize();

    particles_update_gpu_kernel<<<grid, block>>>(m_count, particles, m_dt);
    hipDeviceSynchronize();

    particles_limit_ranges_gpu_kernel<<<grid, block>>>(m_count, particles, m_dt);
    hipDeviceSynchronize();

  #else
    for (unsigned int i = 0; i < m_count; i++)
      particles_main_cpu_kernel(i, m_count, particles, m_dt);

    for (unsigned int i = 0; i < m_count; i++)
      particles_update_cpu_kernel(i, particles, m_dt);

    for (unsigned int i = 0; i < m_count; i++)
      particles_limit_ranges_cpu_kernel(i, particles, m_dt);
  #endif

  m_iteration++;
}

void Particles::set_random()
{
  for (unsigned int i = 0; i < m_count; i++)
  {
    sParticle tmp = random_particle(i);

    #ifdef PARTICLES_USE_CUDA
    hipMemcpy(&particles[i], &tmp, sizeof(sParticle), hipMemcpyHostToDevice);
    #else
    particles[i] = tmp;
    #endif
  }
}

sParticle Particles::random_particle(unsigned int idx)
{
  sParticle result;

  bool plane_mode = true;

  unsigned int side_size;
  float x, y, z;

  if (plane_mode)
  {
    side_size = sqrt(m_count);

    x = idx%side_size;
    y = idx/side_size;
    z = 0.0;

    x = (x/(1.0*side_size) - 0.5)*2.0;
    y = (y/(1.0*side_size) - 0.5)*2.0;
  }
  else
  {
    side_size = cbrt(m_count);

    x = idx%side_size;
    y = (idx/side_size)%side_size;
    z = idx/(side_size*side_size);

    x = (x/(1.0*side_size) - 0.5)*2.0;
    y = (y/(1.0*side_size) - 0.5)*2.0;
    z = (z/(1.0*side_size) - 0.5)*2.0;
  }


  result.r = m_size;
  result.m = 1.0;

  float k = (0.5/side_size);

  result.x = 0.98*(x + k*rnd());
  result.y = 0.98*(y + k*rnd());

  if (plane_mode)
    result.z = 0.0;
  else
    result.z = 0.98*(z + k*rnd());

  result.vx = rnd();
  result.vy = rnd();

  if (plane_mode)
    result.vz = 0.0;
  else
    result.vz = rnd();

  result.ax = 0.0;
  result.ay = 0.0;
  result.az = 0.0;



  return result;
}

float Particles::rnd()
{
  return ((rand()%2000000) - 1000000)/1000000.0;
}
